#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "../include/Ejercicio2GPU.cuh"

#define THR_PER_BLOCK 1024 

__global__ void matrixMulAdd(float *A, float *B, float *C, float *D, int a, int b, int c) {
    //int row = blockIdx.y * blockDim.y + threadIdx.y;
    //int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int col = idx / b;
        int row = idx % b;

    if(idx < a * c) {
        float sum = 0;
        for(int i = 0; i < b; i++) {
            sum += A[row * b + i] * B[i * c + col];
        }
        D[row * c + col] = sum + C[row * c + col];
    }
}


 double matrix_mul_and_add_gpu(int a, int b, int c, float *A, float *B, float *C, float *D) {
    
    hipEvent_t start, stop;
    float *d_A, *d_B, *d_C, *d_D;
    float miliseconds = 0;
    int thr_per_blk, blk_in_grid;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    gpuErrchk(hipMalloc(&d_A, a * b * sizeof(float)));
    gpuErrchk(hipMalloc(&d_B, b * c * sizeof(float)));
    gpuErrchk(hipMalloc(&d_C, a * c * sizeof(float)));
    gpuErrchk(hipMalloc(&d_D, a * c * sizeof(float)));

    gpuErrchk(hipMemcpy(d_A, A, a * b * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, b * c * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C, C, a * c * sizeof(float), hipMemcpyHostToDevice));

    thr_per_blk = THR_PER_BLOCK;
    blk_in_grid = ceil((float)(a * c) / thr_per_blk);

    gpuErrchk(hipEventRecord(start));
    matrixMulAdd<<<blk_in_grid,thr_per_blk>>>(d_A, d_B, d_C, d_D, a, b, c);
    gpuErrchk(hipEventRecord(stop));
   
    hipMemcpy(D, d_D, a * c * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds, start, stop);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return (miliseconds);
}

Tiempo de la CPU: 5.583517Tiempo de la GPU: 48.012383

