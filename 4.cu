#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define THR_PER_BLOCK 1024 
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16
#define WARP_SIZE 32

__global__ void matrixMulAdd(float *A, float *B, float *C, float *D, int a, int b, int c) {
     int warpId = threadIdx.x / WARP_SIZE;


    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> d_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    
    for (int i = 0; i < b; i += WMMA_K) {

        wmma::load_matrix_sync(a_frag, A + (blockIdx.x * WMMA_M + warpId * WMMA_M/2) * b + i, b);
        wmma::load_matrix_sync(b_frag, B + i * c + blockIdx.y * WMMA_N, c);


        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
 wmma::load_matrix_sync(d_frag, C + blockIdx.x * WMMA_M * c + blockIdx.y * WMMA_N, c, wmma::mem_row_major);

    
    for (int i = 0; i < c_frag.num_elements; i++) {
        c_frag.x[i] = c_frag.x[i] + d_frag.x[i];
    }

   
    wmma::store_matrix_sync(D + blockIdx.x * WMMA_M * c + blockIdx.y * WMMA_N, c_frag, c, wmma::mem_row_major);
}


 double matrix_mul_and_add_gpu(int a, int b, int c, float *A, float *B, float *C, float *D) {
    
    hipEvent_t start, stop;
    float *d_A, *d_B, *d_C, *d_D;
    float miliseconds = 0;
    int thr_per_blk, blk_in_grid;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    gpuErrchk(hipMalloc(&d_A, a * b * sizeof(float)));
    gpuErrchk(hipMalloc(&d_B, b * c * sizeof(float)));
    gpuErrchk(hipMalloc(&d_C, a * c * sizeof(float)));
    gpuErrchk(hipMalloc(&d_D, a * c * sizeof(float)));

    gpuErrchk(hipMemcpy(d_A, A, a * b * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, b * c * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C, C, a * c * sizeof(float), hipMemcpyHostToDevice));

    thr_per_blk = THR_PER_BLOCK;
    blk_in_grid = ceil ((float)(a*c) / thr_per_blk);

   gpuErrchk(hipEventRecord(start));
    matrixMulAdd<<<blk_in_grid, thr_per_blk>>>(d_A, d_B, d_C, d_D, a, b, c);
    gpuErrchk(hipEventRecord(stop));
   
    hipMemcpy(D, d_D, a * c * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return (miliseconds);
}



