#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "../include/Ejercicio2.cuh"

#define THR_PER_BLOCK 1024 

__global__ void matrixMulAdd(float *A, float *B, float *C, float *D, int a, int b, int c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < a && col < c) {
        float sum = 0;
        for(int i = 0; i < b; i++) {
            sum += A[row * b + i] * B[i * c + col];
        }
        D[row * c + col] = sum + C[row * c + col];
    }
}


 double matrix_mul_and_add_gpu(int a, int b, int c, float *A, float *B, float *C, float *D) {
    
    hipEvent_t start, stop;
    float *d_A, *d_B, *d_C, *d_D;
    float miliseconds = 0;
    int thr_per_blk, blk_in_grid_x, blk_in_grid_y;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    gpuErrchk(hipMalloc(&d_A, a * b * sizeof(float)));
    gpuErrchk(hipMalloc(&d_B, b * c * sizeof(float)));
    gpuErrchk(hipMalloc(&d_C, a * c * sizeof(float)));
    gpuErrchk(hipMalloc(&d_D, a * c * sizeof(float)));

    gpuErrchk(hipMemcpy(d_A, A, a * b * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, b * c * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C, C, a * c * sizeof(float), hipMemcpyHostToDevice));

    thr_per_blk = THR_PER_BLOCK;
    blk_in_grid_x = (c + thr_per_blk - 1) / thr_per_blk;
    blk_in_grid_y = (a + thr_per_blk - 1) / thr_per_blk;

    gpuErrchk(hipEventRecord(start));
    matrixMulAdd<<<dim3(blk_in_grid_x, blk_in_grid_y), dim3(thr_per_blk, thr_per_blk)>>>(d_A, d_B, d_C, d_D, a, b, c);
    gpuErrchk(hipEventRecord(stop));
   
    hipMemcpy(D, d_D, a * c * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return (miliseconds);
}