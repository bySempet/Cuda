#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "../include/Ejercicio2.cuh"

#define THR_PER_BLOCK 256  

__global__ void matrixMulAdd(float *A, float *B, float *C, float *D, int a, int b, int c) {
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;
   int i;

    
  if(row<a && col<c)
  {
   float sum = 0;
   for(i = 0; i < b; i++)
   {
        sum += A[row * b + i] * B[i * c + col];
    }
    D[row * c + col] = sum + C[row * c + col];
       }
}


double matrix_mul_and_add_gpu(int a, int b, int c, float *A, float *B, float *C, float *D) {
    
   hipEvent_t start, stop;
   float *d_A, *d_B, *d_C, *d_D;
   float miliseconds = 0;
   int thr_per_blk, blk_in_grid;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   gpuErrchk(hipMalloc((void **)&d_A, a * b * sizeof(float)));
   gpuErrchk(hipMalloc((void **)&d_B, b * c * sizeof(float)));
   gpuErrchk(hipMalloc((void **)&d_C, a * c * sizeof(float)));
   gpuErrchk(hipMalloc((void **)&d_D, a * c * sizeof(float)));

   gpuErrchk(hipMemcpy((void *)d_A, A, a * b * sizeof(float), hipMemcpyHostToDevice));
   gpuErrchk(hipMemcpy((void *)d_B, B, b * c * sizeof(float), hipMemcpyHostToDevice));
   gpuErrchk(hipMemcpy((void *)d_C, C, a * c * sizeof(float), hipMemcpyHostToDevice));
   
   dim3 blockDim(16,16);
   dim3 gridDim((c + blockDim.x -1) / blockDim.x,(a + blockDim.y -1)/ blockDim.y);

   gpuErrchk(hipEventRecord(start));
   matrixMulAdd<<<gridDim,blockDim>>>(d_A, d_B, d_C, d_D, a, b, c);
   gpuErrchk(hipEventRecord(stop));
   
   hipMemcpy(D, d_D, a * c * sizeof(float), hipMemcpyDeviceToHost);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&miliseconds, start, stop);
    
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   hipFree(d_D);

   return (miliseconds);
}
